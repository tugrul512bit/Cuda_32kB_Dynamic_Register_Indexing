#ifndef __CUDACC__
#define __CUDACC__
#endif

#include <hip/hip_runtime.h>


#include <cuda_device_runtime_api.h>
#include <hip/device_functions.h>
#include <iostream>
#include <chrono>
using jump_func_t = void(*)(void const*);
template<class F>
jump_func_t jump_func() {
    return [](void const* ptr) { (*static_cast<F const*>(ptr))(); };
}
template<class...Fs>
void jump_table(std::size_t i, Fs const&...fs) {
    struct entry {
        jump_func_t f;
        void const* data;
        void operator()()const { f(data); }
    };
    const entry table[] = {
      {jump_func<Fs>(), std::addressof(fs)}...
    };
    table[i]();
}

__device__ unsigned int getIndex0(unsigned int* arr) { return arr[0]; }
__device__ unsigned int getIndex1(unsigned int* arr) { return arr[1]; }
__device__ unsigned int getIndex2(unsigned int* arr) { return arr[2]; }
__device__ unsigned int getIndex3(unsigned int* arr) { return arr[3]; }
__device__ unsigned int getIndex4(unsigned int* arr) { return arr[4]; }
__device__ unsigned int getIndex5(unsigned int* arr) { return arr[5]; }
__device__ unsigned int getIndex6(unsigned int* arr) { return arr[6]; }
__device__ unsigned int getIndex7(unsigned int* arr) { return arr[7]; }
__device__ unsigned int getIndex8(unsigned int* arr) { return arr[8]; }
__device__ unsigned int getIndex9(unsigned int* arr) { return arr[9]; }

__device__ unsigned int getIndex10(unsigned int* arr) { return arr[10]; }
__device__ unsigned int getIndex11(unsigned int* arr) { return arr[11]; }
__device__ unsigned int getIndex12(unsigned int* arr) { return arr[12]; }
__device__ unsigned int getIndex13(unsigned int* arr) { return arr[13]; }
__device__ unsigned int getIndex14(unsigned int* arr) { return arr[14]; }
__device__ unsigned int getIndex15(unsigned int* arr) { return arr[15]; }
__device__ unsigned int getIndex16(unsigned int* arr) { return arr[16]; }
__device__ unsigned int getIndex17(unsigned int* arr) { return arr[17]; }
__device__ unsigned int getIndex18(unsigned int* arr) { return arr[18]; }
__device__ unsigned int getIndex19(unsigned int* arr) { return arr[19]; }

__device__ unsigned int getIndex20(unsigned int* arr) { return arr[20]; }
__device__ unsigned int getIndex21(unsigned int* arr) { return arr[21]; }
__device__ unsigned int getIndex22(unsigned int* arr) { return arr[22]; }
__device__ unsigned int getIndex23(unsigned int* arr) { return arr[23]; }
__device__ unsigned int getIndex24(unsigned int* arr) { return arr[24]; }
__device__ unsigned int getIndex25(unsigned int* arr) { return arr[25]; }
__device__ unsigned int getIndex26(unsigned int* arr) { return arr[26]; }
__device__ unsigned int getIndex27(unsigned int* arr) { return arr[27]; }
__device__ unsigned int getIndex28(unsigned int* arr) { return arr[28]; }
__device__ unsigned int getIndex29(unsigned int* arr) { return arr[29]; }


__device__ unsigned int getIndex30(unsigned int* arr) { return arr[30]; }
__device__ unsigned int getIndex31(unsigned int* arr) { return arr[31]; }
__device__ unsigned int getIndex32(unsigned int* arr) { return arr[32]; }
__device__ unsigned int getIndex33(unsigned int* arr) { return arr[33]; }
__device__ unsigned int getIndex34(unsigned int* arr) { return arr[34]; }
__device__ unsigned int getIndex35(unsigned int* arr) { return arr[35]; }
__device__ unsigned int getIndex36(unsigned int* arr) { return arr[36]; }
__device__ unsigned int getIndex37(unsigned int* arr) { return arr[37]; }
__device__ unsigned int getIndex38(unsigned int* arr) { return arr[38]; }
__device__ unsigned int getIndex39(unsigned int* arr) { return arr[39]; }


__device__ unsigned int getIndex40(unsigned int* arr) { return arr[40]; }
__device__ unsigned int getIndex41(unsigned int* arr) { return arr[41]; }
__device__ unsigned int getIndex42(unsigned int* arr) { return arr[42]; }
__device__ unsigned int getIndex43(unsigned int* arr) { return arr[43]; }
__device__ unsigned int getIndex44(unsigned int* arr) { return arr[44]; }
__device__ unsigned int getIndex45(unsigned int* arr) { return arr[45]; }
__device__ unsigned int getIndex46(unsigned int* arr) { return arr[46]; }
__device__ unsigned int getIndex47(unsigned int* arr) { return arr[47]; }
__device__ unsigned int getIndex48(unsigned int* arr) { return arr[48]; }
__device__ unsigned int getIndex49(unsigned int* arr) { return arr[49]; }

__device__ unsigned int getIndex50(unsigned int* arr) { return arr[50]; }
__device__ unsigned int getIndex51(unsigned int* arr) { return arr[51]; }
__device__ unsigned int getIndex52(unsigned int* arr) { return arr[52]; }
__device__ unsigned int getIndex53(unsigned int* arr) { return arr[53]; }
__device__ unsigned int getIndex54(unsigned int* arr) { return arr[54]; }
__device__ unsigned int getIndex55(unsigned int* arr) { return arr[55]; }
__device__ unsigned int getIndex56(unsigned int* arr) { return arr[56]; }
__device__ unsigned int getIndex57(unsigned int* arr) { return arr[57]; }
__device__ unsigned int getIndex58(unsigned int* arr) { return arr[58]; }
__device__ unsigned int getIndex59(unsigned int* arr) { return arr[59]; }


__device__ unsigned int getIndex60(unsigned int* arr) { return arr[60]; }
__device__ unsigned int getIndex61(unsigned int* arr) { return arr[61]; }
__device__ unsigned int getIndex62(unsigned int* arr) { return arr[62]; }
__device__ unsigned int getIndex63(unsigned int* arr) { return arr[63]; }
__device__ unsigned int getIndex64(unsigned int* arr) { return arr[64]; }
__device__ unsigned int getIndex65(unsigned int* arr) { return arr[65]; }
__device__ unsigned int getIndex66(unsigned int* arr) { return arr[66]; }
__device__ unsigned int getIndex67(unsigned int* arr) { return arr[67]; }
__device__ unsigned int getIndex68(unsigned int* arr) { return arr[68]; }
__device__ unsigned int getIndex69(unsigned int* arr) { return arr[69]; }


__device__ unsigned int getIndex70(unsigned int* arr) { return arr[70]; }
__device__ unsigned int getIndex71(unsigned int* arr) { return arr[71]; }
__device__ unsigned int getIndex72(unsigned int* arr) { return arr[72]; }
__device__ unsigned int getIndex73(unsigned int* arr) { return arr[73]; }
__device__ unsigned int getIndex74(unsigned int* arr) { return arr[74]; }
__device__ unsigned int getIndex75(unsigned int* arr) { return arr[75]; }
__device__ unsigned int getIndex76(unsigned int* arr) { return arr[76]; }
__device__ unsigned int getIndex77(unsigned int* arr) { return arr[77]; }
__device__ unsigned int getIndex78(unsigned int* arr) { return arr[78]; }
__device__ unsigned int getIndex79(unsigned int* arr) { return arr[79]; }

__device__ unsigned int getIndex80(unsigned int* arr) { return arr[80]; }
__device__ unsigned int getIndex81(unsigned int* arr) { return arr[81]; }
__device__ unsigned int getIndex82(unsigned int* arr) { return arr[82]; }
__device__ unsigned int getIndex83(unsigned int* arr) { return arr[83]; }
__device__ unsigned int getIndex84(unsigned int* arr) { return arr[84]; }
__device__ unsigned int getIndex85(unsigned int* arr) { return arr[85]; }
__device__ unsigned int getIndex86(unsigned int* arr) { return arr[86]; }
__device__ unsigned int getIndex87(unsigned int* arr) { return arr[87]; }
__device__ unsigned int getIndex88(unsigned int* arr) { return arr[88]; }
__device__ unsigned int getIndex89(unsigned int* arr) { return arr[89]; }

__device__ unsigned int getIndex90(unsigned int* arr) { return arr[90]; }
__device__ unsigned int getIndex91(unsigned int* arr) { return arr[91]; }
__device__ unsigned int getIndex92(unsigned int* arr) { return arr[92]; }
__device__ unsigned int getIndex93(unsigned int* arr) { return arr[93]; }
__device__ unsigned int getIndex94(unsigned int* arr) { return arr[94]; }
__device__ unsigned int getIndex95(unsigned int* arr) { return arr[95]; }
__device__ unsigned int getIndex96(unsigned int* arr) { return arr[96]; }
__device__ unsigned int getIndex97(unsigned int* arr) { return arr[97]; }
__device__ unsigned int getIndex98(unsigned int* arr) { return arr[98]; }
__device__ unsigned int getIndex99(unsigned int* arr) { return arr[99]; }

__device__ unsigned int getIndex100(unsigned int* arr) { return arr[100]; }
__device__ unsigned int getIndex101(unsigned int* arr) { return arr[101]; }
__device__ unsigned int getIndex102(unsigned int* arr) { return arr[102]; }
__device__ unsigned int getIndex103(unsigned int* arr) { return arr[103]; }
__device__ unsigned int getIndex104(unsigned int* arr) { return arr[104]; }
__device__ unsigned int getIndex105(unsigned int* arr) { return arr[105]; }
__device__ unsigned int getIndex106(unsigned int* arr) { return arr[106]; }
__device__ unsigned int getIndex107(unsigned int* arr) { return arr[107]; }
__device__ unsigned int getIndex108(unsigned int* arr) { return arr[108]; }
__device__ unsigned int getIndex109(unsigned int* arr) { return arr[109]; }

__device__ unsigned int getIndex110(unsigned int* arr) { return arr[110]; }
__device__ unsigned int getIndex111(unsigned int* arr) { return arr[111]; }
__device__ unsigned int getIndex112(unsigned int* arr) { return arr[112]; }
__device__ unsigned int getIndex113(unsigned int* arr) { return arr[113]; }
__device__ unsigned int getIndex114(unsigned int* arr) { return arr[114]; }
__device__ unsigned int getIndex115(unsigned int* arr) { return arr[115]; }
__device__ unsigned int getIndex116(unsigned int* arr) { return arr[116]; }
__device__ unsigned int getIndex117(unsigned int* arr) { return arr[117]; }
__device__ unsigned int getIndex118(unsigned int* arr) { return arr[118]; }
__device__ unsigned int getIndex119(unsigned int* arr) { return arr[119]; }


__device__ unsigned int getIndex120(unsigned int* arr) { return arr[120]; }
__device__ unsigned int getIndex121(unsigned int* arr) { return arr[121]; }
__device__ unsigned int getIndex122(unsigned int* arr) { return arr[122]; }
__device__ unsigned int getIndex123(unsigned int* arr) { return arr[123]; }
__device__ unsigned int getIndex124(unsigned int* arr) { return arr[124]; }
__device__ unsigned int getIndex125(unsigned int* arr) { return arr[125]; }
__device__ unsigned int getIndex126(unsigned int* arr) { return arr[126]; }
__device__ unsigned int getIndex127(unsigned int* arr) { return arr[127]; }
__device__ unsigned int getIndex128(unsigned int* arr) { return arr[128]; }
__device__ unsigned int getIndex129(unsigned int* arr) { return arr[129]; }

__device__ unsigned int getIndex130(unsigned int* arr) { return arr[130]; }
__device__ unsigned int getIndex131(unsigned int* arr) { return arr[131]; }
__device__ unsigned int getIndex132(unsigned int* arr) { return arr[132]; }
__device__ unsigned int getIndex133(unsigned int* arr) { return arr[133]; }
__device__ unsigned int getIndex134(unsigned int* arr) { return arr[134]; }
__device__ unsigned int getIndex135(unsigned int* arr) { return arr[135]; }
__device__ unsigned int getIndex136(unsigned int* arr) { return arr[136]; }
__device__ unsigned int getIndex137(unsigned int* arr) { return arr[137]; }
__device__ unsigned int getIndex138(unsigned int* arr) { return arr[138]; }
__device__ unsigned int getIndex139(unsigned int* arr) { return arr[139]; }

__device__ unsigned int getIndex140(unsigned int* arr) { return arr[140]; }
__device__ unsigned int getIndex141(unsigned int* arr) { return arr[141]; }
__device__ unsigned int getIndex142(unsigned int* arr) { return arr[142]; }
__device__ unsigned int getIndex143(unsigned int* arr) { return arr[143]; }
__device__ unsigned int getIndex144(unsigned int* arr) { return arr[144]; }
__device__ unsigned int getIndex145(unsigned int* arr) { return arr[145]; }
__device__ unsigned int getIndex146(unsigned int* arr) { return arr[146]; }
__device__ unsigned int getIndex147(unsigned int* arr) { return arr[147]; }
__device__ unsigned int getIndex148(unsigned int* arr) { return arr[148]; }
__device__ unsigned int getIndex149(unsigned int* arr) { return arr[149]; }

__device__ unsigned int getIndex150(unsigned int* arr) { return arr[150]; }
__device__ unsigned int getIndex151(unsigned int* arr) { return arr[151]; }
__device__ unsigned int getIndex152(unsigned int* arr) { return arr[152]; }
__device__ unsigned int getIndex153(unsigned int* arr) { return arr[153]; }
__device__ unsigned int getIndex154(unsigned int* arr) { return arr[154]; }
__device__ unsigned int getIndex155(unsigned int* arr) { return arr[155]; }
__device__ unsigned int getIndex156(unsigned int* arr) { return arr[156]; }
__device__ unsigned int getIndex157(unsigned int* arr) { return arr[157]; }
__device__ unsigned int getIndex158(unsigned int* arr) { return arr[158]; }
__device__ unsigned int getIndex159(unsigned int* arr) { return arr[159]; }

__device__ unsigned int getIndex160(unsigned int* arr) { return arr[160]; }
__device__ unsigned int getIndex161(unsigned int* arr) { return arr[161]; }
__device__ unsigned int getIndex162(unsigned int* arr) { return arr[162]; }
__device__ unsigned int getIndex163(unsigned int* arr) { return arr[163]; }
__device__ unsigned int getIndex164(unsigned int* arr) { return arr[164]; }
__device__ unsigned int getIndex165(unsigned int* arr) { return arr[165]; }
__device__ unsigned int getIndex166(unsigned int* arr) { return arr[166]; }
__device__ unsigned int getIndex167(unsigned int* arr) { return arr[167]; }
__device__ unsigned int getIndex168(unsigned int* arr) { return arr[168]; }
__device__ unsigned int getIndex169(unsigned int* arr) { return arr[169]; }

__device__ unsigned int getIndex170(unsigned int* arr) { return arr[170]; }
__device__ unsigned int getIndex171(unsigned int* arr) { return arr[171]; }
__device__ unsigned int getIndex172(unsigned int* arr) { return arr[172]; }
__device__ unsigned int getIndex173(unsigned int* arr) { return arr[173]; }
__device__ unsigned int getIndex174(unsigned int* arr) { return arr[174]; }
__device__ unsigned int getIndex175(unsigned int* arr) { return arr[175]; }
__device__ unsigned int getIndex176(unsigned int* arr) { return arr[176]; }
__device__ unsigned int getIndex177(unsigned int* arr) { return arr[177]; }
__device__ unsigned int getIndex178(unsigned int* arr) { return arr[178]; }
__device__ unsigned int getIndex179(unsigned int* arr) { return arr[179]; }

__device__ unsigned int getIndex180(unsigned int* arr) { return arr[180]; }
__device__ unsigned int getIndex181(unsigned int* arr) { return arr[181]; }
__device__ unsigned int getIndex182(unsigned int* arr) { return arr[182]; }
__device__ unsigned int getIndex183(unsigned int* arr) { return arr[183]; }
__device__ unsigned int getIndex184(unsigned int* arr) { return arr[184]; }
__device__ unsigned int getIndex185(unsigned int* arr) { return arr[185]; }
__device__ unsigned int getIndex186(unsigned int* arr) { return arr[186]; }
__device__ unsigned int getIndex187(unsigned int* arr) { return arr[187]; }
__device__ unsigned int getIndex188(unsigned int* arr) { return arr[188]; }
__device__ unsigned int getIndex189(unsigned int* arr) { return arr[189]; }

__device__ unsigned int getIndex190(unsigned int* arr) { return arr[190]; }
__device__ unsigned int getIndex191(unsigned int* arr) { return arr[191]; }
__device__ unsigned int getIndex192(unsigned int* arr) { return arr[192]; }
__device__ unsigned int getIndex193(unsigned int* arr) { return arr[193]; }
__device__ unsigned int getIndex194(unsigned int* arr) { return arr[194]; }
__device__ unsigned int getIndex195(unsigned int* arr) { return arr[195]; }
__device__ unsigned int getIndex196(unsigned int* arr) { return arr[196]; }
__device__ unsigned int getIndex197(unsigned int* arr) { return arr[197]; }
__device__ unsigned int getIndex198(unsigned int* arr) { return arr[198]; }
__device__ unsigned int getIndex199(unsigned int* arr) { return arr[199]; }


__device__ unsigned int getIndex200(unsigned int* arr) { return arr[200]; }
__device__ unsigned int getIndex201(unsigned int* arr) { return arr[201]; }
__device__ unsigned int getIndex202(unsigned int* arr) { return arr[202]; }
__device__ unsigned int getIndex203(unsigned int* arr) { return arr[203]; }
__device__ unsigned int getIndex204(unsigned int* arr) { return arr[204]; }
__device__ unsigned int getIndex205(unsigned int* arr) { return arr[205]; }
__device__ unsigned int getIndex206(unsigned int* arr) { return arr[206]; }
__device__ unsigned int getIndex207(unsigned int* arr) { return arr[207]; }
__device__ unsigned int getIndex208(unsigned int* arr) { return arr[208]; }
__device__ unsigned int getIndex209(unsigned int* arr) { return arr[209]; }

__device__ unsigned int getIndex210(unsigned int* arr) { return arr[210]; }
__device__ unsigned int getIndex211(unsigned int* arr) { return arr[211]; }
__device__ unsigned int getIndex212(unsigned int* arr) { return arr[212]; }
__device__ unsigned int getIndex213(unsigned int* arr) { return arr[213]; }
__device__ unsigned int getIndex214(unsigned int* arr) { return arr[214]; }
__device__ unsigned int getIndex215(unsigned int* arr) { return arr[215]; }
__device__ unsigned int getIndex216(unsigned int* arr) { return arr[216]; }
__device__ unsigned int getIndex217(unsigned int* arr) { return arr[217]; }
__device__ unsigned int getIndex218(unsigned int* arr) { return arr[218]; }
__device__ unsigned int getIndex219(unsigned int* arr) { return arr[219]; }

__device__ unsigned int getIndex220(unsigned int* arr) { return arr[220]; }
__device__ unsigned int getIndex221(unsigned int* arr) { return arr[221]; }
__device__ unsigned int getIndex222(unsigned int* arr) { return arr[222]; }
__device__ unsigned int getIndex223(unsigned int* arr) { return arr[223]; }
__device__ unsigned int getIndex224(unsigned int* arr) { return arr[224]; }
__device__ unsigned int getIndex225(unsigned int* arr) { return arr[225]; }
__device__ unsigned int getIndex226(unsigned int* arr) { return arr[226]; }
__device__ unsigned int getIndex227(unsigned int* arr) { return arr[227]; }
__device__ unsigned int getIndex228(unsigned int* arr) { return arr[228]; }
__device__ unsigned int getIndex229(unsigned int* arr) { return arr[229]; }

__device__ unsigned int getIndex230(unsigned int* arr) { return arr[230]; }
__device__ unsigned int getIndex231(unsigned int* arr) { return arr[231]; }
__device__ unsigned int getIndex232(unsigned int* arr) { return arr[232]; }
__device__ unsigned int getIndex233(unsigned int* arr) { return arr[233]; }
__device__ unsigned int getIndex234(unsigned int* arr) { return arr[234]; }
__device__ unsigned int getIndex235(unsigned int* arr) { return arr[235]; }
__device__ unsigned int getIndex236(unsigned int* arr) { return arr[236]; }
__device__ unsigned int getIndex237(unsigned int* arr) { return arr[237]; }
__device__ unsigned int getIndex238(unsigned int* arr) { return arr[238]; }
__device__ unsigned int getIndex239(unsigned int* arr) { return arr[239]; }

__device__ unsigned int getIndex240(unsigned int* arr) { return arr[240]; }
__device__ unsigned int getIndex241(unsigned int* arr) { return arr[241]; }
__device__ unsigned int getIndex242(unsigned int* arr) { return arr[242]; }
__device__ unsigned int getIndex243(unsigned int* arr) { return arr[243]; }
__device__ unsigned int getIndex244(unsigned int* arr) { return arr[244]; }
__device__ unsigned int getIndex245(unsigned int* arr) { return arr[245]; }
__device__ unsigned int getIndex246(unsigned int* arr) { return arr[246]; }
__device__ unsigned int getIndex247(unsigned int* arr) { return arr[247]; }
__device__ unsigned int getIndex248(unsigned int* arr) { return arr[248]; }
__device__ unsigned int getIndex249(unsigned int* arr) { return arr[249]; }

__device__ unsigned int getIndex250(unsigned int* arr) { return arr[250]; }
__device__ unsigned int getIndex251(unsigned int* arr) { return arr[251]; }
__device__ unsigned int getIndex252(unsigned int* arr) { return arr[252]; }
__device__ unsigned int getIndex253(unsigned int* arr) { return arr[253]; }
__device__ unsigned int getIndex254(unsigned int* arr) { return arr[254]; }
__device__ unsigned int getIndex255(unsigned int* arr) { return arr[255]; }


__device__ void setIndex0(unsigned int* arr, unsigned int data) { arr[0]=data; }
__device__ void setIndex1(unsigned int* arr, unsigned int data) { arr[1]=data; }
__device__ void setIndex2(unsigned int* arr, unsigned int data) { arr[2]=data; }
__device__ void setIndex3(unsigned int* arr, unsigned int data) { arr[3]=data; }
__device__ void setIndex4(unsigned int* arr, unsigned int data) { arr[4]=data; }
__device__ void setIndex5(unsigned int* arr, unsigned int data) { arr[5]=data; }
__device__ void setIndex6(unsigned int* arr, unsigned int data) { arr[6]=data; }
__device__ void setIndex7(unsigned int* arr, unsigned int data) { arr[7]=data; }
__device__ void setIndex8(unsigned int* arr, unsigned int data) { arr[8] = data; }
__device__ void setIndex9(unsigned int* arr, unsigned int data) { arr[9] = data; }

__device__ void setIndex10(unsigned int* arr, unsigned int data) { arr[10] = data; }
__device__ void setIndex11(unsigned int* arr, unsigned int data) { arr[11] = data; }
__device__ void setIndex12(unsigned int* arr, unsigned int data) { arr[12] = data; }
__device__ void setIndex13(unsigned int* arr, unsigned int data) { arr[13] = data; }
__device__ void setIndex14(unsigned int* arr, unsigned int data) { arr[14] = data; }
__device__ void setIndex15(unsigned int* arr, unsigned int data) { arr[15] = data; }
__device__ void setIndex16(unsigned int* arr, unsigned int data) { arr[16] = data; }
__device__ void setIndex17(unsigned int* arr, unsigned int data) { arr[17] = data; }
__device__ void setIndex18(unsigned int* arr, unsigned int data) { arr[18] = data; }
__device__ void setIndex19(unsigned int* arr, unsigned int data) { arr[19] = data; }

__device__ void setIndex20(unsigned int* arr, unsigned int data) { arr[20] = data; }
__device__ void setIndex21(unsigned int* arr, unsigned int data) { arr[21] = data; }
__device__ void setIndex22(unsigned int* arr, unsigned int data) { arr[22] = data; }
__device__ void setIndex23(unsigned int* arr, unsigned int data) { arr[23] = data; }
__device__ void setIndex24(unsigned int* arr, unsigned int data) { arr[24] = data; }
__device__ void setIndex25(unsigned int* arr, unsigned int data) { arr[25] = data; }
__device__ void setIndex26(unsigned int* arr, unsigned int data) { arr[26] = data; }
__device__ void setIndex27(unsigned int* arr, unsigned int data) { arr[27] = data; }
__device__ void setIndex28(unsigned int* arr, unsigned int data) { arr[28] = data; }
__device__ void setIndex29(unsigned int* arr, unsigned int data) { arr[29] = data; }

__device__ void setIndex30(unsigned int* arr, unsigned int data) { arr[30] = data; }
__device__ void setIndex31(unsigned int* arr, unsigned int data) { arr[31] = data; }
__device__ void setIndex32(unsigned int* arr, unsigned int data) { arr[32] = data; }
__device__ void setIndex33(unsigned int* arr, unsigned int data) { arr[33] = data; }
__device__ void setIndex34(unsigned int* arr, unsigned int data) { arr[34] = data; }
__device__ void setIndex35(unsigned int* arr, unsigned int data) { arr[35] = data; }
__device__ void setIndex36(unsigned int* arr, unsigned int data) { arr[36] = data; }
__device__ void setIndex37(unsigned int* arr, unsigned int data) { arr[37] = data; }
__device__ void setIndex38(unsigned int* arr, unsigned int data) { arr[38] = data; }
__device__ void setIndex39(unsigned int* arr, unsigned int data) { arr[39] = data; }

__device__ void setIndex40(unsigned int* arr, unsigned int data) { arr[40] = data; }
__device__ void setIndex41(unsigned int* arr, unsigned int data) { arr[41] = data; }
__device__ void setIndex42(unsigned int* arr, unsigned int data) { arr[42] = data; }
__device__ void setIndex43(unsigned int* arr, unsigned int data) { arr[43] = data; }
__device__ void setIndex44(unsigned int* arr, unsigned int data) { arr[44] = data; }
__device__ void setIndex45(unsigned int* arr, unsigned int data) { arr[45] = data; }
__device__ void setIndex46(unsigned int* arr, unsigned int data) { arr[46] = data; }
__device__ void setIndex47(unsigned int* arr, unsigned int data) { arr[47] = data; }
__device__ void setIndex48(unsigned int* arr, unsigned int data) { arr[48] = data; }
__device__ void setIndex49(unsigned int* arr, unsigned int data) { arr[49] = data; }

__device__ void setIndex50(unsigned int* arr, unsigned int data) { arr[50] = data; }
__device__ void setIndex51(unsigned int* arr, unsigned int data) { arr[51] = data; }
__device__ void setIndex52(unsigned int* arr, unsigned int data) { arr[52] = data; }
__device__ void setIndex53(unsigned int* arr, unsigned int data) { arr[53] = data; }
__device__ void setIndex54(unsigned int* arr, unsigned int data) { arr[54] = data; }
__device__ void setIndex55(unsigned int* arr, unsigned int data) { arr[55] = data; }
__device__ void setIndex56(unsigned int* arr, unsigned int data) { arr[56] = data; }
__device__ void setIndex57(unsigned int* arr, unsigned int data) { arr[57] = data; }
__device__ void setIndex58(unsigned int* arr, unsigned int data) { arr[58] = data; }
__device__ void setIndex59(unsigned int* arr, unsigned int data) { arr[59] = data; }

__device__ void setIndex60(unsigned int* arr, unsigned int data) { arr[60] = data; }
__device__ void setIndex61(unsigned int* arr, unsigned int data) { arr[61] = data; }
__device__ void setIndex62(unsigned int* arr, unsigned int data) { arr[62] = data; }
__device__ void setIndex63(unsigned int* arr, unsigned int data) { arr[63] = data; }
__device__ void setIndex64(unsigned int* arr, unsigned int data) { arr[64] = data; }
__device__ void setIndex65(unsigned int* arr, unsigned int data) { arr[65] = data; }
__device__ void setIndex66(unsigned int* arr, unsigned int data) { arr[66] = data; }
__device__ void setIndex67(unsigned int* arr, unsigned int data) { arr[67] = data; }
__device__ void setIndex68(unsigned int* arr, unsigned int data) { arr[68] = data; }
__device__ void setIndex69(unsigned int* arr, unsigned int data) { arr[69] = data; }

__device__ void setIndex70(unsigned int* arr, unsigned int data) { arr[70] = data; }
__device__ void setIndex71(unsigned int* arr, unsigned int data) { arr[71] = data; }
__device__ void setIndex72(unsigned int* arr, unsigned int data) { arr[72] = data; }
__device__ void setIndex73(unsigned int* arr, unsigned int data) { arr[73] = data; }
__device__ void setIndex74(unsigned int* arr, unsigned int data) { arr[74] = data; }
__device__ void setIndex75(unsigned int* arr, unsigned int data) { arr[75] = data; }
__device__ void setIndex76(unsigned int* arr, unsigned int data) { arr[76] = data; }
__device__ void setIndex77(unsigned int* arr, unsigned int data) { arr[77] = data; }
__device__ void setIndex78(unsigned int* arr, unsigned int data) { arr[78] = data; }
__device__ void setIndex79(unsigned int* arr, unsigned int data) { arr[79] = data; }

__device__ void setIndex80(unsigned int* arr, unsigned int data) { arr[80] = data; }
__device__ void setIndex81(unsigned int* arr, unsigned int data) { arr[81] = data; }
__device__ void setIndex82(unsigned int* arr, unsigned int data) { arr[82] = data; }
__device__ void setIndex83(unsigned int* arr, unsigned int data) { arr[83] = data; }
__device__ void setIndex84(unsigned int* arr, unsigned int data) { arr[84] = data; }
__device__ void setIndex85(unsigned int* arr, unsigned int data) { arr[85] = data; }
__device__ void setIndex86(unsigned int* arr, unsigned int data) { arr[86] = data; }
__device__ void setIndex87(unsigned int* arr, unsigned int data) { arr[87] = data; }
__device__ void setIndex88(unsigned int* arr, unsigned int data) { arr[88] = data; }
__device__ void setIndex89(unsigned int* arr, unsigned int data) { arr[89] = data; }

__device__ void setIndex90(unsigned int* arr, unsigned int data) { arr[90] = data; }
__device__ void setIndex91(unsigned int* arr, unsigned int data) { arr[91] = data; }
__device__ void setIndex92(unsigned int* arr, unsigned int data) { arr[92] = data; }
__device__ void setIndex93(unsigned int* arr, unsigned int data) { arr[93] = data; }
__device__ void setIndex94(unsigned int* arr, unsigned int data) { arr[94] = data; }
__device__ void setIndex95(unsigned int* arr, unsigned int data) { arr[95] = data; }
__device__ void setIndex96(unsigned int* arr, unsigned int data) { arr[96] = data; }
__device__ void setIndex97(unsigned int* arr, unsigned int data) { arr[97] = data; }
__device__ void setIndex98(unsigned int* arr, unsigned int data) { arr[98] = data; }
__device__ void setIndex99(unsigned int* arr, unsigned int data) { arr[99] = data; }

__device__ void setIndex100(unsigned int* arr, unsigned int data) { arr[100] = data; }
__device__ void setIndex101(unsigned int* arr, unsigned int data) { arr[101] = data; }
__device__ void setIndex102(unsigned int* arr, unsigned int data) { arr[102] = data; }
__device__ void setIndex103(unsigned int* arr, unsigned int data) { arr[103] = data; }
__device__ void setIndex104(unsigned int* arr, unsigned int data) { arr[104] = data; }
__device__ void setIndex105(unsigned int* arr, unsigned int data) { arr[105] = data; }
__device__ void setIndex106(unsigned int* arr, unsigned int data) { arr[106] = data; }
__device__ void setIndex107(unsigned int* arr, unsigned int data) { arr[107] = data; }
__device__ void setIndex108(unsigned int* arr, unsigned int data) { arr[108] = data; }
__device__ void setIndex109(unsigned int* arr, unsigned int data) { arr[109] = data; }

__device__ void setIndex110(unsigned int* arr, unsigned int data) { arr[110] = data; }
__device__ void setIndex111(unsigned int* arr, unsigned int data) { arr[111] = data; }
__device__ void setIndex112(unsigned int* arr, unsigned int data) { arr[112] = data; }
__device__ void setIndex113(unsigned int* arr, unsigned int data) { arr[113] = data; }
__device__ void setIndex114(unsigned int* arr, unsigned int data) { arr[114] = data; }
__device__ void setIndex115(unsigned int* arr, unsigned int data) { arr[115] = data; }
__device__ void setIndex116(unsigned int* arr, unsigned int data) { arr[116] = data; }
__device__ void setIndex117(unsigned int* arr, unsigned int data) { arr[117] = data; }
__device__ void setIndex118(unsigned int* arr, unsigned int data) { arr[118] = data; }
__device__ void setIndex119(unsigned int* arr, unsigned int data) { arr[119] = data; }

__device__ void setIndex120(unsigned int* arr, unsigned int data) { arr[120] = data; }
__device__ void setIndex121(unsigned int* arr, unsigned int data) { arr[121] = data; }
__device__ void setIndex122(unsigned int* arr, unsigned int data) { arr[122] = data; }
__device__ void setIndex123(unsigned int* arr, unsigned int data) { arr[123] = data; }
__device__ void setIndex124(unsigned int* arr, unsigned int data) { arr[124] = data; }
__device__ void setIndex125(unsigned int* arr, unsigned int data) { arr[125] = data; }
__device__ void setIndex126(unsigned int* arr, unsigned int data) { arr[126] = data; }
__device__ void setIndex127(unsigned int* arr, unsigned int data) { arr[127] = data; }
__device__ void setIndex128(unsigned int* arr, unsigned int data) { arr[128] = data; }
__device__ void setIndex129(unsigned int* arr, unsigned int data) { arr[129] = data; }

__device__ void setIndex130(unsigned int* arr, unsigned int data) { arr[130] = data; }
__device__ void setIndex131(unsigned int* arr, unsigned int data) { arr[131] = data; }
__device__ void setIndex132(unsigned int* arr, unsigned int data) { arr[132] = data; }
__device__ void setIndex133(unsigned int* arr, unsigned int data) { arr[133] = data; }
__device__ void setIndex134(unsigned int* arr, unsigned int data) { arr[134] = data; }
__device__ void setIndex135(unsigned int* arr, unsigned int data) { arr[135] = data; }
__device__ void setIndex136(unsigned int* arr, unsigned int data) { arr[136] = data; }
__device__ void setIndex137(unsigned int* arr, unsigned int data) { arr[137] = data; }
__device__ void setIndex138(unsigned int* arr, unsigned int data) { arr[138] = data; }
__device__ void setIndex139(unsigned int* arr, unsigned int data) { arr[139] = data; }

__device__ void setIndex140(unsigned int* arr, unsigned int data) { arr[140] = data; }
__device__ void setIndex141(unsigned int* arr, unsigned int data) { arr[141] = data; }
__device__ void setIndex142(unsigned int* arr, unsigned int data) { arr[142] = data; }
__device__ void setIndex143(unsigned int* arr, unsigned int data) { arr[143] = data; }
__device__ void setIndex144(unsigned int* arr, unsigned int data) { arr[144] = data; }
__device__ void setIndex145(unsigned int* arr, unsigned int data) { arr[145] = data; }
__device__ void setIndex146(unsigned int* arr, unsigned int data) { arr[146] = data; }
__device__ void setIndex147(unsigned int* arr, unsigned int data) { arr[147] = data; }
__device__ void setIndex148(unsigned int* arr, unsigned int data) { arr[148] = data; }
__device__ void setIndex149(unsigned int* arr, unsigned int data) { arr[149] = data; }


__device__ void setIndex150(unsigned int* arr, unsigned int data) { arr[150] = data; }
__device__ void setIndex151(unsigned int* arr, unsigned int data) { arr[151] = data; }
__device__ void setIndex152(unsigned int* arr, unsigned int data) { arr[152] = data; }
__device__ void setIndex153(unsigned int* arr, unsigned int data) { arr[153] = data; }
__device__ void setIndex154(unsigned int* arr, unsigned int data) { arr[154] = data; }
__device__ void setIndex155(unsigned int* arr, unsigned int data) { arr[155] = data; }
__device__ void setIndex156(unsigned int* arr, unsigned int data) { arr[156] = data; }
__device__ void setIndex157(unsigned int* arr, unsigned int data) { arr[157] = data; }
__device__ void setIndex158(unsigned int* arr, unsigned int data) { arr[158] = data; }
__device__ void setIndex159(unsigned int* arr, unsigned int data) { arr[159] = data; }


__device__ void setIndex160(unsigned int* arr, unsigned int data) { arr[160] = data; }
__device__ void setIndex161(unsigned int* arr, unsigned int data) { arr[161] = data; }
__device__ void setIndex162(unsigned int* arr, unsigned int data) { arr[162] = data; }
__device__ void setIndex163(unsigned int* arr, unsigned int data) { arr[163] = data; }
__device__ void setIndex164(unsigned int* arr, unsigned int data) { arr[164] = data; }
__device__ void setIndex165(unsigned int* arr, unsigned int data) { arr[165] = data; }
__device__ void setIndex166(unsigned int* arr, unsigned int data) { arr[166] = data; }
__device__ void setIndex167(unsigned int* arr, unsigned int data) { arr[167] = data; }
__device__ void setIndex168(unsigned int* arr, unsigned int data) { arr[168] = data; }
__device__ void setIndex169(unsigned int* arr, unsigned int data) { arr[169] = data; }

__device__ void setIndex170(unsigned int* arr, unsigned int data) { arr[170] = data; }
__device__ void setIndex171(unsigned int* arr, unsigned int data) { arr[171] = data; }
__device__ void setIndex172(unsigned int* arr, unsigned int data) { arr[172] = data; }
__device__ void setIndex173(unsigned int* arr, unsigned int data) { arr[173] = data; }
__device__ void setIndex174(unsigned int* arr, unsigned int data) { arr[174] = data; }
__device__ void setIndex175(unsigned int* arr, unsigned int data) { arr[175] = data; }
__device__ void setIndex176(unsigned int* arr, unsigned int data) { arr[176] = data; }
__device__ void setIndex177(unsigned int* arr, unsigned int data) { arr[177] = data; }
__device__ void setIndex178(unsigned int* arr, unsigned int data) { arr[178] = data; }
__device__ void setIndex179(unsigned int* arr, unsigned int data) { arr[179] = data; }

__device__ void setIndex180(unsigned int* arr, unsigned int data) { arr[180] = data; }
__device__ void setIndex181(unsigned int* arr, unsigned int data) { arr[181] = data; }
__device__ void setIndex182(unsigned int* arr, unsigned int data) { arr[182] = data; }
__device__ void setIndex183(unsigned int* arr, unsigned int data) { arr[183] = data; }
__device__ void setIndex184(unsigned int* arr, unsigned int data) { arr[184] = data; }
__device__ void setIndex185(unsigned int* arr, unsigned int data) { arr[185] = data; }
__device__ void setIndex186(unsigned int* arr, unsigned int data) { arr[186] = data; }
__device__ void setIndex187(unsigned int* arr, unsigned int data) { arr[187] = data; }
__device__ void setIndex188(unsigned int* arr, unsigned int data) { arr[188] = data; }
__device__ void setIndex189(unsigned int* arr, unsigned int data) { arr[189] = data; }

__device__ void setIndex190(unsigned int* arr, unsigned int data) { arr[190] = data; }
__device__ void setIndex191(unsigned int* arr, unsigned int data) { arr[191] = data; }
__device__ void setIndex192(unsigned int* arr, unsigned int data) { arr[192] = data; }
__device__ void setIndex193(unsigned int* arr, unsigned int data) { arr[193] = data; }
__device__ void setIndex194(unsigned int* arr, unsigned int data) { arr[194] = data; }
__device__ void setIndex195(unsigned int* arr, unsigned int data) { arr[195] = data; }
__device__ void setIndex196(unsigned int* arr, unsigned int data) { arr[196] = data; }
__device__ void setIndex197(unsigned int* arr, unsigned int data) { arr[197] = data; }
__device__ void setIndex198(unsigned int* arr, unsigned int data) { arr[198] = data; }
__device__ void setIndex199(unsigned int* arr, unsigned int data) { arr[199] = data; }

__device__ void setIndex200(unsigned int* arr, unsigned int data) { arr[200] = data; }
__device__ void setIndex201(unsigned int* arr, unsigned int data) { arr[201] = data; }
__device__ void setIndex202(unsigned int* arr, unsigned int data) { arr[202] = data; }
__device__ void setIndex203(unsigned int* arr, unsigned int data) { arr[203] = data; }
__device__ void setIndex204(unsigned int* arr, unsigned int data) { arr[204] = data; }
__device__ void setIndex205(unsigned int* arr, unsigned int data) { arr[205] = data; }
__device__ void setIndex206(unsigned int* arr, unsigned int data) { arr[206] = data; }
__device__ void setIndex207(unsigned int* arr, unsigned int data) { arr[207] = data; }
__device__ void setIndex208(unsigned int* arr, unsigned int data) { arr[208] = data; }
__device__ void setIndex209(unsigned int* arr, unsigned int data) { arr[209] = data; }

__device__ void setIndex210(unsigned int* arr, unsigned int data) { arr[210] = data; }
__device__ void setIndex211(unsigned int* arr, unsigned int data) { arr[211] = data; }
__device__ void setIndex212(unsigned int* arr, unsigned int data) { arr[212] = data; }
__device__ void setIndex213(unsigned int* arr, unsigned int data) { arr[213] = data; }
__device__ void setIndex214(unsigned int* arr, unsigned int data) { arr[214] = data; }
__device__ void setIndex215(unsigned int* arr, unsigned int data) { arr[215] = data; }
__device__ void setIndex216(unsigned int* arr, unsigned int data) { arr[216] = data; }
__device__ void setIndex217(unsigned int* arr, unsigned int data) { arr[217] = data; }
__device__ void setIndex218(unsigned int* arr, unsigned int data) { arr[218] = data; }
__device__ void setIndex219(unsigned int* arr, unsigned int data) { arr[219] = data; }

__device__ void setIndex220(unsigned int* arr, unsigned int data) { arr[220] = data; }
__device__ void setIndex221(unsigned int* arr, unsigned int data) { arr[221] = data; }
__device__ void setIndex222(unsigned int* arr, unsigned int data) { arr[222] = data; }
__device__ void setIndex223(unsigned int* arr, unsigned int data) { arr[223] = data; }
__device__ void setIndex224(unsigned int* arr, unsigned int data) { arr[224] = data; }
__device__ void setIndex225(unsigned int* arr, unsigned int data) { arr[225] = data; }
__device__ void setIndex226(unsigned int* arr, unsigned int data) { arr[226] = data; }
__device__ void setIndex227(unsigned int* arr, unsigned int data) { arr[227] = data; }
__device__ void setIndex228(unsigned int* arr, unsigned int data) { arr[228] = data; }
__device__ void setIndex229(unsigned int* arr, unsigned int data) { arr[229] = data; }

__device__ void setIndex230(unsigned int* arr, unsigned int data) { arr[230] = data; }
__device__ void setIndex231(unsigned int* arr, unsigned int data) { arr[231] = data; }
__device__ void setIndex232(unsigned int* arr, unsigned int data) { arr[232] = data; }
__device__ void setIndex233(unsigned int* arr, unsigned int data) { arr[233] = data; }
__device__ void setIndex234(unsigned int* arr, unsigned int data) { arr[234] = data; }
__device__ void setIndex235(unsigned int* arr, unsigned int data) { arr[235] = data; }
__device__ void setIndex236(unsigned int* arr, unsigned int data) { arr[236] = data; }
__device__ void setIndex237(unsigned int* arr, unsigned int data) { arr[237] = data; }
__device__ void setIndex238(unsigned int* arr, unsigned int data) { arr[238] = data; }
__device__ void setIndex239(unsigned int* arr, unsigned int data) { arr[239] = data; }

__device__ void setIndex240(unsigned int* arr, unsigned int data) { arr[240] = data; }
__device__ void setIndex241(unsigned int* arr, unsigned int data) { arr[241] = data; }
__device__ void setIndex242(unsigned int* arr, unsigned int data) { arr[242] = data; }
__device__ void setIndex243(unsigned int* arr, unsigned int data) { arr[243] = data; }
__device__ void setIndex244(unsigned int* arr, unsigned int data) { arr[244] = data; }
__device__ void setIndex245(unsigned int* arr, unsigned int data) { arr[245] = data; }
__device__ void setIndex246(unsigned int* arr, unsigned int data) { arr[246] = data; }
__device__ void setIndex247(unsigned int* arr, unsigned int data) { arr[247] = data; }
__device__ void setIndex248(unsigned int* arr, unsigned int data) { arr[248] = data; }
__device__ void setIndex249(unsigned int* arr, unsigned int data) { arr[249] = data; }

__device__ void setIndex250(unsigned int* arr, unsigned int data) { arr[250] = data; }
__device__ void setIndex251(unsigned int* arr, unsigned int data) { arr[251] = data; }
__device__ void setIndex252(unsigned int* arr, unsigned int data) { arr[252] = data; }
__device__ void setIndex253(unsigned int* arr, unsigned int data) { arr[253] = data; }
__device__ void setIndex254(unsigned int* arr, unsigned int data) { arr[254] = data; }
__device__ void setIndex255(unsigned int* arr, unsigned int data) { arr[255] = data; }


typedef unsigned int (*funcGet) (unsigned int*);
typedef void (*funcSet) (unsigned int*,unsigned int);
__device__ funcGet fuGet[256] = { 
    &getIndex0,&getIndex1,&getIndex2,&getIndex3,&getIndex4,&getIndex5,&getIndex6,&getIndex7,&getIndex8,&getIndex9,
    &getIndex10,&getIndex11,&getIndex12,&getIndex13,&getIndex14,&getIndex15,&getIndex16,&getIndex17,&getIndex18,&getIndex19,
    &getIndex20,&getIndex21,&getIndex22,&getIndex23,&getIndex24,&getIndex25,&getIndex26,&getIndex27,&getIndex28,&getIndex29,
    & getIndex30,& getIndex31,& getIndex32,& getIndex33,& getIndex34,& getIndex35,& getIndex36,& getIndex37,& getIndex38,& getIndex39,
    & getIndex40,& getIndex41,& getIndex42,& getIndex43,& getIndex44,& getIndex45,& getIndex46,& getIndex47,& getIndex48,& getIndex49,
    & getIndex50,& getIndex51,& getIndex52,& getIndex53,& getIndex54,& getIndex55,& getIndex56,& getIndex57,& getIndex58,& getIndex59,
    & getIndex60,& getIndex61,& getIndex62,& getIndex63,& getIndex64,& getIndex65,& getIndex66,& getIndex67,& getIndex68,& getIndex69,
    & getIndex70,& getIndex71,& getIndex72,& getIndex73,& getIndex74,& getIndex75,& getIndex76,& getIndex77,& getIndex78,& getIndex79,
    & getIndex80,& getIndex81,& getIndex82,& getIndex83,& getIndex84,& getIndex85,& getIndex86,& getIndex87,& getIndex88,& getIndex89,
    & getIndex90,& getIndex91,& getIndex92,& getIndex93,& getIndex94,& getIndex95,& getIndex96,& getIndex97,& getIndex98,& getIndex99,

    & getIndex100,& getIndex101,& getIndex102,& getIndex103,& getIndex104,& getIndex105,& getIndex106,& getIndex107,& getIndex108,& getIndex109,
    & getIndex110,& getIndex111,& getIndex112,& getIndex113,& getIndex114,& getIndex115,& getIndex116,& getIndex117,& getIndex118,& getIndex119,
    & getIndex120,& getIndex121,& getIndex122,& getIndex123,& getIndex124,& getIndex125,& getIndex126,& getIndex127,& getIndex128,& getIndex129,
    & getIndex130,& getIndex131,& getIndex132,& getIndex133,& getIndex134,& getIndex135,& getIndex136,& getIndex137,& getIndex138,& getIndex139,
    & getIndex140,& getIndex141,& getIndex142,& getIndex143,& getIndex144,& getIndex145,& getIndex146,& getIndex147,& getIndex148,& getIndex149,
    & getIndex150,& getIndex151,& getIndex152,& getIndex153,& getIndex154,& getIndex155,& getIndex156,& getIndex157,& getIndex158,& getIndex159,
    & getIndex160,& getIndex161,& getIndex162,& getIndex163,& getIndex164,& getIndex165,& getIndex166,& getIndex167,& getIndex168,& getIndex169,
    & getIndex170,& getIndex171,& getIndex172,& getIndex173,& getIndex174,& getIndex175,& getIndex176,& getIndex177,& getIndex178,& getIndex179,
    & getIndex180,& getIndex181,& getIndex182,& getIndex183,& getIndex184,& getIndex185,& getIndex186,& getIndex187,& getIndex188,& getIndex189,
    & getIndex190,& getIndex191,& getIndex192,& getIndex193,& getIndex194,& getIndex195,& getIndex196,& getIndex197,& getIndex198,& getIndex199,

    & getIndex200,& getIndex201,& getIndex202,& getIndex203,& getIndex204,& getIndex205,& getIndex206,& getIndex207,& getIndex208,& getIndex209,
    & getIndex200,& getIndex211,& getIndex212,& getIndex213,& getIndex214,& getIndex215,& getIndex216,& getIndex217,& getIndex218,& getIndex219,
    & getIndex200,& getIndex221,& getIndex222,& getIndex223,& getIndex224,& getIndex225,& getIndex226,& getIndex227,& getIndex228,& getIndex229,
    & getIndex200,& getIndex231,& getIndex232,& getIndex233,& getIndex234,& getIndex235,& getIndex236,& getIndex237,& getIndex238,& getIndex239,
    & getIndex200,& getIndex241,& getIndex242,& getIndex243,& getIndex244,& getIndex245,& getIndex246,& getIndex247,& getIndex248,& getIndex249,
    & getIndex200,& getIndex251,& getIndex252,& getIndex253,& getIndex254,& getIndex255
};
__device__ funcSet fuSet[256] = { 
    &setIndex0,&setIndex1,&setIndex2,&setIndex3,&setIndex4,&setIndex5,&setIndex6,&setIndex7,&setIndex8,&setIndex9,
    &setIndex10,&setIndex11,&setIndex12,&setIndex13,&setIndex14,&setIndex15,&setIndex16,&setIndex17,&setIndex18,&setIndex19,
    &setIndex20,&setIndex21,&setIndex22,&setIndex23,&setIndex24,&setIndex25,&setIndex26,&setIndex27,&setIndex28,&setIndex29,
    & setIndex30,& setIndex31,& setIndex32,& setIndex33,& setIndex34,& setIndex35,& setIndex36,& setIndex37,& setIndex38,& setIndex39,
    & setIndex40,& setIndex41,& setIndex42,& setIndex43,& setIndex44,& setIndex45,& setIndex46,& setIndex47,& setIndex48,& setIndex49,
    & setIndex50,& setIndex51,& setIndex52,& setIndex53,& setIndex54,& setIndex55,& setIndex56,& setIndex57,& setIndex58,& setIndex59,
    & setIndex60,& setIndex61,& setIndex62,& setIndex63,& setIndex64,& setIndex65,& setIndex66,& setIndex67,& setIndex68,& setIndex69,
    & setIndex70,& setIndex71,& setIndex72,& setIndex73,& setIndex74,& setIndex75,& setIndex76,& setIndex77,& setIndex78,& setIndex79,
    & setIndex80,& setIndex81,& setIndex82,& setIndex83,& setIndex84,& setIndex85,& setIndex86,& setIndex87,& setIndex88,& setIndex89,
    & setIndex90,& setIndex91,& setIndex92,& setIndex93,& setIndex94,& setIndex95,& setIndex96,& setIndex97,& setIndex98,& setIndex99,

    & setIndex100,& setIndex101,& setIndex102,& setIndex103,& setIndex104,& setIndex105,& setIndex106,& setIndex107,& setIndex108,& setIndex109,
    & setIndex110,& setIndex111,& setIndex112,& setIndex113,& setIndex114,& setIndex115,& setIndex116,& setIndex117,& setIndex118,& setIndex119,
    & setIndex120,& setIndex121,& setIndex122,& setIndex123,& setIndex124,& setIndex125,& setIndex126,& setIndex127,& setIndex128,& setIndex129,
    & setIndex130,& setIndex131,& setIndex132,& setIndex133,& setIndex134,& setIndex135,& setIndex136,& setIndex137,& setIndex138,& setIndex139,
    & setIndex140,& setIndex141,& setIndex142,& setIndex143,& setIndex144,& setIndex145,& setIndex146,& setIndex147,& setIndex148,& setIndex149,
    & setIndex150,& setIndex151,& setIndex152,& setIndex153,& setIndex154,& setIndex155,& setIndex156,& setIndex157,& setIndex158,& setIndex159,
    & setIndex160,& setIndex161,& setIndex162,& setIndex163,& setIndex164,& setIndex165,& setIndex166,& setIndex167,& setIndex168,& setIndex169,
    & setIndex170,& setIndex171,& setIndex172,& setIndex173,& setIndex174,& setIndex175,& setIndex176,& setIndex177,& setIndex178,& setIndex179,
    & setIndex180,& setIndex181,& setIndex182,& setIndex183,& setIndex184,& setIndex185,& setIndex186,& setIndex187,& setIndex188,& setIndex189,
    & setIndex190,& setIndex191,& setIndex192,& setIndex193,& setIndex194,& setIndex195,& setIndex196,& setIndex197,& setIndex198,& setIndex199,

    & setIndex200,& setIndex201,& setIndex202,& setIndex203,& setIndex204,& setIndex205,& setIndex206,& setIndex207,& setIndex208,& setIndex209,
    & setIndex210,& setIndex211,& setIndex212,& setIndex213,& setIndex214,& setIndex215,& setIndex216,& setIndex217,& setIndex218,& setIndex219,
    & setIndex220,& setIndex221,& setIndex222,& setIndex223,& setIndex224,& setIndex225,& setIndex226,& setIndex227,& setIndex228,& setIndex229,
    & setIndex230,& setIndex231,& setIndex232,& setIndex233,& setIndex234,& setIndex235,& setIndex236,& setIndex237,& setIndex238,& setIndex239,
    & setIndex240,& setIndex241,& setIndex242,& setIndex243,& setIndex244,& setIndex245,& setIndex246,& setIndex247,& setIndex248,& setIndex249,
    & setIndex250,& setIndex251,& setIndex252,& setIndex253,& setIndex254,& setIndex255
};

template<int ArraySize>
struct WarpRegisterArray
{
private:
    unsigned int mem[(1 + (ArraySize - 1) / 32)];

    // main thread broadcasts index
    __device__ int broadcastIndexFromMainThread(const unsigned int mask, int index)
    {
        return __shfl_sync(mask, index, 0);
    }

    // main thread broadcasts data (to set)
    __device__ int broadcastDataFromMainThread(const unsigned int mask, int index, int data)
    {
        return __shfl_sync(mask, data, 0);
    }

    // main thread knows where the data has to come from
    __device__ unsigned int gatherData(const unsigned int mask, int data, int row)
    {
        return __shfl_sync(mask, data, row);
    }

    

public:
    __device__ unsigned int get(const int index)
    {
        const int id = threadIdx.x;        
        constexpr unsigned int mask = 0xffffffff;
        const int indexReceived = broadcastIndexFromMainThread(mask, index);
        const int rowReceived = indexReceived / (1 + (ArraySize - 1) / 32);
        int result = 0;
        if (rowReceived == id)
        {
            const int column = indexReceived % (1 + (ArraySize - 1) / 32);
            result = fuGet[column](mem);
           
        }
        // main thread computes the right lane without need to receive
        return gatherData(mask, result, rowReceived);
    }
    __device__ void set(const unsigned int data, const int index)
    {
        const int id = threadIdx.x;
        constexpr unsigned int mask = 0xffffffff;
        const int indexReceived = broadcastIndexFromMainThread(mask, index);
        const int dataReceived = broadcastIndexFromMainThread(mask, data);
        const int rowReceived = indexReceived / (1 + (ArraySize - 1) / 32);
        int result = 0;
        if (rowReceived == id)
        {
            const int column = indexReceived % (1 + (ArraySize - 1) / 32);
            fuSet[column](mem,dataReceived);
       
        }
    }
};


__global__ void dynamicRegisterIndexing(int* result, int start, int stop)
{
    WarpRegisterArray<8160> arr;
    for (int j = 0; j < 100; j++)
    {
        int sum = 0;

        for (int i = start; i < stop; i++)
            arr.set(1, i);

        for (int i = start; i < stop; i++)
        {
            auto data = arr.get(i);
            sum += data;
        }

        if (threadIdx.x == 0)
            result[0] += sum;
    }
}


int main()
{
 
    int* data;
    hipMallocManaged(&data, sizeof(int));
    int start, stop;
    std::cin >> start;
    std::cin >> stop;
    *data = 0;
    for (int i = 0; i < 10; i++)
    {
        dynamicRegisterIndexing <<<1, 32 >>> (data, start, stop);
        hipDeviceSynchronize();
    }
    std::cout << "sum  = " << *data << std::endl;
    hipFree(data);
    return 0;
}
