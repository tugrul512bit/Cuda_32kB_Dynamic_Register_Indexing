#ifndef __CUDACC__
#define __CUDACC__
#endif

#include <hip/hip_runtime.h>


#include <cuda_device_runtime_api.h>
#include <hip/device_functions.h>
#include <iostream>
#include <chrono>

template<int ArraySize>
struct WarpRegisterArray
{
private:
    int mem[(1 + (ArraySize - 1) / 32)];

    // main thread broadcasts index
    __device__ int broadcastIndexFromMainThread(const unsigned int mask, int index)
    {
        return __shfl_sync(mask, index, 0);
    }

    // main thread broadcasts data (to set)
    __device__ int broadcastDataFromMainThread(const unsigned int mask, int index, int data)
    {
        return __shfl_sync(mask, data, 0);
    }

    // main thread knows where the data has to come from
    __device__ unsigned int gatherData(const unsigned int mask, int data, int row)
    {
        return __shfl_sync(mask, data, row);
    }



public:
    __device__ unsigned int get(const int index)
    {
        const int id = threadIdx.x;        
        constexpr unsigned int mask = 0xffffffff;
        const int indexReceived = broadcastIndexFromMainThread(mask, index);
        const int rowReceived = indexReceived / (1 + (ArraySize - 1) / 32);
        int result = 0;
        if (rowReceived == id)
        {
            const int column = indexReceived % (1 + (ArraySize - 1) / 32);
            switch (column)
            {
            case 0: result = mem[0]; break;
            case 1: result = mem[1]; break;
            case 2: result = mem[2]; break;
            case 3: result = mem[3]; break;
            case 4: result = mem[4]; break;
            case 5: result = mem[5]; break;
            case 6: result = mem[6]; break;
            case 7: result = mem[7]; break;
            case 8: result = mem[8]; break;
            case 9: result = mem[9]; break;
            case 10: result = mem[10]; break;
            case 11: result = mem[11]; break;
            case 12: result = mem[12]; break;
            case 13: result = mem[13]; break;
            case 14: result = mem[14]; break;
            case 15: result = mem[15]; break;
            case 16: result = mem[16]; break;
            case 17: result = mem[17]; break;
            case 18: result = mem[18]; break;
            case 19: result = mem[19]; break;
            case 20: result = mem[20]; break;
            case 21: result = mem[21]; break;
            case 22: result = mem[22]; break;
            case 23: result = mem[23]; break;
            case 24: result = mem[24]; break;
            case 25: result = mem[25]; break;
            case 26: result = mem[26]; break;
            case 27: result = mem[27]; break;
            case 28: result = mem[28]; break;
            case 29: result = mem[29]; break;
            case 30: result = mem[30]; break;
            case 31: result = mem[31]; break;
            case 32: result = mem[32]; break;
            case 33: result = mem[33]; break;
            case 34: result = mem[34]; break;
            case 35: result = mem[35]; break;
            case 36: result = mem[36]; break;
            case 37: result = mem[37]; break;
            case 38: result = mem[38]; break;
            case 39: result = mem[39]; break;
            case 40: result = mem[40]; break;
            case 41: result = mem[41]; break;
            case 42: result = mem[42]; break;
            case 43: result = mem[43]; break;
            case 44: result = mem[44]; break;
            case 45: result = mem[45]; break;
            case 46: result = mem[46]; break;
            case 47: result = mem[47]; break;
            case 48: result = mem[48]; break;
            case 49: result = mem[49]; break;
            case 50: result = mem[50]; break;
            case 51: result = mem[51]; break;
            case 52: result = mem[52]; break;
            case 53: result = mem[53]; break;
            case 54: result = mem[54]; break;
            case 55: result = mem[55]; break;
            case 56: result = mem[56]; break;
            case 57: result = mem[57]; break;
            case 58: result = mem[58]; break;
            case 59: result = mem[59]; break;
            case 60: result = mem[60]; break;
            case 61: result = mem[61]; break;
            case 62: result = mem[62]; break;
            case 63: result = mem[63]; break;
            case 64: result = mem[64]; break;
            case 65: result = mem[65]; break;
            case 66: result = mem[66]; break;
            case 67: result = mem[67]; break;
            case 68: result = mem[68]; break;
            case 69: result = mem[69]; break;
            case 70: result = mem[70]; break;
            case 71: result = mem[71]; break;
            case 72: result = mem[72]; break;
            case 73: result = mem[73]; break;
            case 74: result = mem[74]; break;
            case 75: result = mem[75]; break;
            case 76: result = mem[76]; break;
            case 77: result = mem[77]; break;
            case 78: result = mem[78]; break;
            case 79: result = mem[79]; break;
            case 80: result = mem[80]; break;
            case 81: result = mem[81]; break;
            case 82: result = mem[82]; break;
            case 83: result = mem[83]; break;
            case 84: result = mem[84]; break;
            case 85: result = mem[85]; break;
            case 86: result = mem[86]; break;
            case 87: result = mem[87]; break;
            case 88: result = mem[88]; break;
            case 89: result = mem[89]; break;
            case 90: result = mem[90]; break;
            case 91: result = mem[91]; break;
            case 92: result = mem[92]; break;
            case 93: result = mem[93]; break;
            case 94: result = mem[94]; break;
            case 95: result = mem[95]; break;
            case 96: result = mem[96]; break;
            case 97: result = mem[97]; break;
            case 98: result = mem[98]; break;
            case 99: result = mem[99]; break;
            case 100: result = mem[100]; break;
            case 101: result = mem[101]; break;
            case 102: result = mem[102]; break;
            case 103: result = mem[103]; break;
            case 104: result = mem[104]; break;
            case 105: result = mem[105]; break;
            case 106: result = mem[106]; break;
            case 107: result = mem[107]; break;
            case 108: result = mem[108]; break;
            case 109: result = mem[109]; break;
            case 110: result = mem[110]; break;
            case 111: result = mem[111]; break;
            case 112: result = mem[112]; break;
            case 113: result = mem[113]; break;
            case 114: result = mem[114]; break;
            case 115: result = mem[115]; break;
            case 116: result = mem[116]; break;
            case 117: result = mem[117]; break;
            case 118: result = mem[118]; break;
            case 119: result = mem[119]; break;
            case 120: result = mem[120]; break;
            case 121: result = mem[121]; break;
            case 122: result = mem[122]; break;
            case 123: result = mem[123]; break;
            case 124: result = mem[124]; break;
            case 125: result = mem[125]; break;
            case 126: result = mem[126]; break;
            case 127: result = mem[127]; break;
            case 128: result = mem[128]; break;
            case 129: result = mem[129]; break;
            case 130: result = mem[130]; break;
            case 131: result = mem[131]; break;
            case 132: result = mem[132]; break;
            case 133: result = mem[133]; break;
            case 134: result = mem[134]; break;
            case 135: result = mem[135]; break;
            case 136: result = mem[136]; break;
            case 137: result = mem[137]; break;
            case 138: result = mem[138]; break;
            case 139: result = mem[139]; break;
            case 140: result = mem[140]; break;
            case 141: result = mem[141]; break;
            case 142: result = mem[142]; break;
            case 143: result = mem[143]; break;
            case 144: result = mem[144]; break;
            case 145: result = mem[145]; break;
            case 146: result = mem[146]; break;
            case 147: result = mem[147]; break;
            case 148: result = mem[148]; break;
            case 149: result = mem[149]; break;
            case 150: result = mem[150]; break;
            case 151: result = mem[151]; break;
            case 152: result = mem[152]; break;
            case 153: result = mem[153]; break;
            case 154: result = mem[154]; break;
            case 155: result = mem[155]; break;
            case 156: result = mem[156]; break;
            case 157: result = mem[157]; break;
            case 158: result = mem[158]; break;
            case 159: result = mem[159]; break;
            case 160: result = mem[160]; break;
            case 161: result = mem[161]; break;
            case 162: result = mem[162]; break;
            case 163: result = mem[163]; break;
            case 164: result = mem[164]; break;
            case 165: result = mem[165]; break;
            case 166: result = mem[166]; break;
            case 167: result = mem[167]; break;
            case 168: result = mem[168]; break;
            case 169: result = mem[169]; break;
            case 170: result = mem[170]; break;
            case 171: result = mem[171]; break;
            case 172: result = mem[172]; break;
            case 173: result = mem[173]; break;
            case 174: result = mem[174]; break;
            case 175: result = mem[175]; break;
            case 176: result = mem[176]; break;
            case 177: result = mem[177]; break;
            case 178: result = mem[178]; break;
            case 179: result = mem[179]; break;
            case 180: result = mem[180]; break;
            case 181: result = mem[181]; break;
            case 182: result = mem[182]; break;
            case 183: result = mem[183]; break;
            case 184: result = mem[184]; break;
            case 185: result = mem[185]; break;
            case 186: result = mem[186]; break;
            case 187: result = mem[187]; break;
            case 188: result = mem[188]; break;
            case 189: result = mem[189]; break;
            case 190: result = mem[190]; break;
            case 191: result = mem[191]; break;
            case 192: result = mem[192]; break;
            case 193: result = mem[193]; break;
            case 194: result = mem[194]; break;
            case 195: result = mem[195]; break;
            case 196: result = mem[196]; break;
            case 197: result = mem[197]; break;
            case 198: result = mem[198]; break;
            case 199: result = mem[199]; break;
            case 200: result = mem[200]; break;
            case 201: result = mem[201]; break;
            case 202: result = mem[202]; break;
            case 203: result = mem[203]; break;
            case 204: result = mem[204]; break;
            case 205: result = mem[205]; break;
            case 206: result = mem[206]; break;
            case 207: result = mem[207]; break;
            case 208: result = mem[208]; break;
            case 209: result = mem[209]; break;
            case 210: result = mem[210]; break;
            case 211: result = mem[211]; break;
            case 212: result = mem[212]; break;
            case 213: result = mem[213]; break;
            case 214: result = mem[214]; break;
            case 215: result = mem[215]; break;
            case 216: result = mem[216]; break;
            case 217: result = mem[217]; break;
            case 218: result = mem[218]; break;
            case 219: result = mem[219]; break;
            case 220: result = mem[220]; break;
            case 221: result = mem[221]; break;
            case 222: result = mem[222]; break;
            case 223: result = mem[223]; break;
            case 224: result = mem[224]; break;
            case 225: result = mem[225]; break;
            case 226: result = mem[226]; break;
            case 227: result = mem[227]; break;
            case 228: result = mem[228]; break;
            case 229: result = mem[229]; break;
            case 230: result = mem[230]; break;
            case 231: result = mem[231]; break;
            case 232: result = mem[232]; break;
            case 233: result = mem[233]; break;
            case 234: result = mem[234]; break;
            case 235: result = mem[235]; break;
            case 236: result = mem[236]; break;
            case 237: result = mem[237]; break;
            case 238: result = mem[238]; break;
            case 239: result = mem[239]; break;
            case 240: result = mem[240]; break;
            case 241: result = mem[241]; break;
            case 242: result = mem[242]; break;
            case 243: result = mem[243]; break;
            case 244: result = mem[244]; break;
            case 245: result = mem[245]; break;
            case 246: result = mem[246]; break;
            case 247: result = mem[247]; break;
            case 248: result = mem[248]; break;
            case 249: result = mem[249]; break;
            case 250: result = mem[250]; break;
            case 251: result = mem[251]; break;
            case 252: result = mem[252]; break;
            case 253: result = mem[253]; break;
            case 254: result = mem[254]; break;
            case 255: result = mem[255]; break;
            default:break;
            }
        }
        // main thread computes the right lane without need to receive
        return gatherData(mask, result, rowReceived);
    }
    __device__ void set(const unsigned int data, const int index)
    {
        const int id = threadIdx.x;
        constexpr unsigned int mask = 0xffffffff;
        const int indexReceived = broadcastIndexFromMainThread(mask, index);
        const int dataReceived = broadcastIndexFromMainThread(mask, data);
        const int rowReceived = indexReceived / (1 + (ArraySize - 1) / 32);
        int result = 0;
        if (rowReceived == id)
        {
            const int column = indexReceived % (1 + (ArraySize - 1) / 32);
            switch (column)
            {
            case 0:  mem[0] = dataReceived; break;
            case 1:  mem[1] = dataReceived; break;
            case 2:  mem[2] = dataReceived; break;
            case 3:  mem[3] = dataReceived; break;
            case 4:  mem[4] = dataReceived; break;
            case 5:  mem[5] = dataReceived; break;
            case 6:  mem[6] = dataReceived; break;
            case 7:  mem[7] = dataReceived; break;
            case 8:  mem[8] = dataReceived; break;
            case 9:  mem[9] = dataReceived; break;
            case 10: mem[10] = dataReceived; break;
            case 11: mem[11] = dataReceived; break;
            case 12: mem[12] = dataReceived; break;
            case 13: mem[13] = dataReceived; break;
            case 14: mem[14] = dataReceived; break;
            case 15: mem[15] = dataReceived; break;
            case 16: mem[16] = dataReceived; break;
            case 17: mem[17] = dataReceived; break;
            case 18: mem[18] = dataReceived; break;
            case 19: mem[19] = dataReceived; break;
            case 20: mem[20] = dataReceived; break;
            case 21: mem[21] = dataReceived; break;
            case 22: mem[22] = dataReceived; break;
            case 23: mem[23] = dataReceived; break;
            case 24: mem[24] = dataReceived; break;
            case 25: mem[25] = dataReceived; break;
            case 26: mem[26] = dataReceived; break;
            case 27: mem[27] = dataReceived; break;
            case 28: mem[28] = dataReceived; break;
            case 29: mem[29] = dataReceived; break;
            case 30: mem[30] = dataReceived; break;
            case 31: mem[31] = dataReceived; break;
            case 32: mem[32] = dataReceived; break;
            case 33: mem[33] = dataReceived; break;
            case 34: mem[34] = dataReceived; break;
            case 35: mem[35] = dataReceived; break;
            case 36: mem[36] = dataReceived; break;
            case 37: mem[37] = dataReceived; break;
            case 38: mem[38] = dataReceived; break;
            case 39: mem[39] = dataReceived; break;
            case 40: mem[40] = dataReceived; break;
            case 41: mem[41] = dataReceived; break;
            case 42: mem[42] = dataReceived; break;
            case 43: mem[43] = dataReceived; break;
            case 44: mem[44] = dataReceived; break;
            case 45: mem[45] = dataReceived; break;
            case 46: mem[46] = dataReceived; break;
            case 47: mem[47] = dataReceived; break;
            case 48: mem[48] = dataReceived; break;
            case 49: mem[49] = dataReceived; break;
            case 50: mem[50] = dataReceived; break;
            case 51: mem[51] = dataReceived; break;
            case 52: mem[52] = dataReceived; break;
            case 53: mem[53] = dataReceived; break;
            case 54: mem[54] = dataReceived; break;
            case 55: mem[55] = dataReceived; break;
            case 56: mem[56] = dataReceived; break;
            case 57: mem[57] = dataReceived; break;
            case 58: mem[58] = dataReceived; break;
            case 59: mem[59] = dataReceived; break;
            case 60: mem[60] = dataReceived; break;
            case 61: mem[61] = dataReceived; break;
            case 62: mem[62] = dataReceived; break;
            case 63: mem[63] = dataReceived; break;
            case 64: mem[64] = dataReceived; break;
            case 65: mem[65] = dataReceived; break;
            case 66: mem[66] = dataReceived; break;
            case 67: mem[67] = dataReceived; break;
            case 68: mem[68] = dataReceived; break;
            case 69: mem[69] = dataReceived; break;
            case 70: mem[70] = dataReceived; break;
            case 71: mem[71] = dataReceived; break;
            case 72: mem[72] = dataReceived; break;
            case 73: mem[73] = dataReceived; break;
            case 74: mem[74] = dataReceived; break;
            case 75: mem[75] = dataReceived; break;
            case 76: mem[76] = dataReceived; break;
            case 77: mem[77] = dataReceived; break;
            case 78: mem[78] = dataReceived; break;
            case 79: mem[79] = dataReceived; break;
            case 80: mem[80] = dataReceived; break;
            case 81: mem[81] = dataReceived; break;
            case 82: mem[82] = dataReceived; break;
            case 83: mem[83] = dataReceived; break;
            case 84: mem[84] = dataReceived; break;
            case 85: mem[85] = dataReceived; break;
            case 86: mem[86] = dataReceived; break;
            case 87: mem[87] = dataReceived; break;
            case 88: mem[88] = dataReceived; break;
            case 89: mem[89] = dataReceived; break;
            case 90: mem[90] = dataReceived; break;
            case 91: mem[91] = dataReceived; break;
            case 92: mem[92] = dataReceived; break;
            case 93: mem[93] = dataReceived; break;
            case 94: mem[94] = dataReceived; break;
            case 95: mem[95] = dataReceived; break;
            case 96: mem[96] = dataReceived; break;
            case 97: mem[97] = dataReceived; break;
            case 98: mem[98] = dataReceived; break;
            case 99: mem[99] = dataReceived; break;
            case 100: mem[100] = dataReceived; break;
            case 101: mem[101] = dataReceived; break;
            case 102: mem[102] = dataReceived; break;
            case 103: mem[103] = dataReceived; break;
            case 104: mem[104] = dataReceived; break;
            case 105: mem[105] = dataReceived; break;
            case 106: mem[106] = dataReceived; break;
            case 107: mem[107] = dataReceived; break;
            case 108: mem[108] = dataReceived; break;
            case 109: mem[109] = dataReceived; break;
            case 110: mem[110] = dataReceived; break;
            case 111: mem[111] = dataReceived; break;
            case 112: mem[112] = dataReceived; break;
            case 113: mem[113] = dataReceived; break;
            case 114: mem[114] = dataReceived; break;
            case 115: mem[115] = dataReceived; break;
            case 116: mem[116] = dataReceived; break;
            case 117: mem[117] = dataReceived; break;
            case 118: mem[118] = dataReceived; break;
            case 119: mem[119] = dataReceived; break;
            case 120: mem[120] = dataReceived; break;
            case 121: mem[121] = dataReceived; break;
            case 122: mem[122] = dataReceived; break;
            case 123: mem[123] = dataReceived; break;
            case 124: mem[124] = dataReceived; break;
            case 125: mem[125] = dataReceived; break;
            case 126: mem[126] = dataReceived; break;
            case 127: mem[127] = dataReceived; break;
            case 128: mem[128] = dataReceived; break;
            case 129: mem[129] = dataReceived; break;
            case 130: mem[130] = dataReceived; break;
            case 131: mem[131] = dataReceived; break;
            case 132: mem[132] = dataReceived; break;
            case 133: mem[133] = dataReceived; break;
            case 134: mem[134] = dataReceived; break;
            case 135: mem[135] = dataReceived; break;
            case 136: mem[136] = dataReceived; break;
            case 137: mem[137] = dataReceived; break;
            case 138: mem[138] = dataReceived; break;
            case 139: mem[139] = dataReceived; break;
            case 140: mem[140] = dataReceived; break;
            case 141: mem[141] = dataReceived; break;
            case 142: mem[142] = dataReceived; break;
            case 143: mem[143] = dataReceived; break;
            case 144: mem[144] = dataReceived; break;
            case 145: mem[145] = dataReceived; break;
            case 146: mem[146] = dataReceived; break;
            case 147: mem[147] = dataReceived; break;
            case 148: mem[148] = dataReceived; break;
            case 149: mem[149] = dataReceived; break;
            case 150: mem[150] = dataReceived; break;
            case 151: mem[151] = dataReceived; break;
            case 152: mem[152] = dataReceived; break;
            case 153: mem[153] = dataReceived; break;
            case 154: mem[154] = dataReceived; break;
            case 155: mem[155] = dataReceived; break;
            case 156: mem[156] = dataReceived; break;
            case 157: mem[157] = dataReceived; break;
            case 158: mem[158] = dataReceived; break;
            case 159: mem[159] = dataReceived; break;
            case 160: mem[160] = dataReceived; break;
            case 161: mem[161] = dataReceived; break;
            case 162: mem[162] = dataReceived; break;
            case 163: mem[163] = dataReceived; break;
            case 164: mem[164] = dataReceived; break;
            case 165: mem[165] = dataReceived; break;
            case 166: mem[166] = dataReceived; break;
            case 167: mem[167] = dataReceived; break;
            case 168: mem[168] = dataReceived; break;
            case 169: mem[169] = dataReceived; break;
            case 170: mem[170] = dataReceived; break;
            case 171: mem[171] = dataReceived; break;
            case 172: mem[172] = dataReceived; break;
            case 173: mem[173] = dataReceived; break;
            case 174: mem[174] = dataReceived; break;
            case 175: mem[175] = dataReceived; break;
            case 176: mem[176] = dataReceived; break;
            case 177: mem[177] = dataReceived; break;
            case 178: mem[178] = dataReceived; break;
            case 179: mem[179] = dataReceived; break;
            case 180: mem[180] = dataReceived; break;
            case 181: mem[181] = dataReceived; break;
            case 182: mem[182] = dataReceived; break;
            case 183: mem[183] = dataReceived; break;
            case 184: mem[184] = dataReceived; break;
            case 185: mem[185] = dataReceived; break;
            case 186: mem[186] = dataReceived; break;
            case 187: mem[187] = dataReceived; break;
            case 188: mem[188] = dataReceived; break;
            case 189: mem[189] = dataReceived; break;
            case 190: mem[190] = dataReceived; break;
            case 191: mem[191] = dataReceived; break;
            case 192: mem[192] = dataReceived; break;
            case 193: mem[193] = dataReceived; break;
            case 194: mem[194] = dataReceived; break;
            case 195: mem[195] = dataReceived; break;
            case 196: mem[196] = dataReceived; break;
            case 197: mem[197] = dataReceived; break;
            case 198: mem[198] = dataReceived; break;
            case 199: mem[199] = dataReceived; break;
            case 200: mem[200] = dataReceived; break;
            case 201: mem[201] = dataReceived; break;
            case 202: mem[202] = dataReceived; break;
            case 203: mem[203] = dataReceived; break;
            case 204: mem[204] = dataReceived; break;
            case 205: mem[205] = dataReceived; break;
            case 206: mem[206] = dataReceived; break;
            case 207: mem[207] = dataReceived; break;
            case 208: mem[208] = dataReceived; break;
            case 209: mem[209] = dataReceived; break;
            case 210: mem[210] = dataReceived; break;
            case 211: mem[211] = dataReceived; break;
            case 212: mem[212] = dataReceived; break;
            case 213: mem[213] = dataReceived; break;
            case 214: mem[214] = dataReceived; break;
            case 215: mem[215] = dataReceived; break;
            case 216: mem[216] = dataReceived; break;
            case 217: mem[217] = dataReceived; break;
            case 218: mem[218] = dataReceived; break;
            case 219: mem[219] = dataReceived; break;
            case 220: mem[220] = dataReceived; break;
            case 221: mem[221] = dataReceived; break;
            case 222: mem[222] = dataReceived; break;
            case 223: mem[223] = dataReceived; break;
            case 224: mem[224] = dataReceived; break;
            case 225: mem[225] = dataReceived; break;
            case 226: mem[226] = dataReceived; break;
            case 227: mem[227] = dataReceived; break;
            case 228: mem[228] = dataReceived; break;
            case 229: mem[229] = dataReceived; break;
            case 230: mem[230] = dataReceived; break;
            case 231: mem[231] = dataReceived; break;
            case 232: mem[232] = dataReceived; break;
            case 233: mem[233] = dataReceived; break;
            case 234: mem[234] = dataReceived; break;
            case 235: mem[235] = dataReceived; break;
            case 236: mem[236] = dataReceived; break;
            case 237: mem[237] = dataReceived; break;
            case 238: mem[238] = dataReceived; break;
            case 239: mem[239] = dataReceived; break;
            case 240: mem[240] = dataReceived; break;
            case 241: mem[241] = dataReceived; break;
            case 242: mem[242] = dataReceived; break;
            case 243: mem[243] = dataReceived; break;
            case 244: mem[244] = dataReceived; break;
            case 245: mem[245] = dataReceived; break;
            case 246: mem[246] = dataReceived; break;
            case 247: mem[247] = dataReceived; break;
            case 248: mem[248] = dataReceived; break;
            case 249: mem[249] = dataReceived; break;
            case 250: mem[250] = dataReceived; break;
            case 251: mem[251] = dataReceived; break;
            case 252: mem[252] = dataReceived; break;
            case 253: mem[253] = dataReceived; break;
            case 254: mem[254] = dataReceived; break;
            case 255: mem[255] = dataReceived; break;
            default:break;
            }
        }
    }
};


__global__ void dynamicRegisterIndexing(int* result, int start, int stop)
{
    WarpRegisterArray<5000> arr;
    int sum = 0;

    for (int i = start; i < stop; i++)
        arr.set(1, i);

    for (int i = start; i < stop; i++)
    {
        auto data = arr.get(i);
        sum += data;
    }

    if (threadIdx.x == 0)
        result[0] = sum;
}


int main()
{
    int* data;
    hipMallocManaged(&data, sizeof(int));
    int start, stop;
    std::cin >> start;
    std::cin >> stop;
    for (int i = 0; i < 10; i++)
    {
        dynamicRegisterIndexing <<<1, 32 >>> (data, start, stop);
        hipDeviceSynchronize();
    }
    std::cout << "sum  = " << *data << std::endl;
    hipFree(data);
    return 0;
}
